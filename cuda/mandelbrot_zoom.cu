#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define WIDTH 1920        
#define HEIGHT 1080      
#define MAX_ITER 5000    

global void computeMandelbrot(double xmin, double xmax, double ymin, double ymax, int width, int height, unsigned char *output) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        double real = xmin + (xmax - xmin) * x / width;
        double imag = ymin + (ymax - ymin) * y / height;

        double z_real = 0.0, z_imag = 0.0;
        int n = 0;

        while (z_real * z_real + z_imag * z_imag <= 4.0 && n < MAX_ITER) {
            double temp_real = z_real * z_real - z_imag * z_imag + real;
            z_imag = 2.0 * z_real * z_imag + imag;
            z_real = temp_real;
            n++;
        }

        double log_zn = logf(z_real * z_real + z_imag * z_imag) / 2.0f; 
        double nu = logf(log_zn / logf(2.0f)) / logf(2.0f);            
        double iter_smooth = n + 1 - nu;

        int color = (int)(255.0 * iter_smooth / MAX_ITER);
        int idx = 3 * (y * width + x); // R, G, B for each pixel
        output[idx] = color;          // Red
        output[idx + 1] = (color * 5) % 255; // Green
        output[idx + 2] = (color * 10) % 255; // Blue
    }
}

void savePPM(const char *filename, unsigned char *data, int width, int height) {
    FILE *fp = fopen(filename, "wb");
    fprintf(fp, "P6\n%d %d\n255\n", width, height); 
    fwrite(data, sizeof(unsigned char), width * height * 3, fp);
    fclose(fp);
}

int main() {
    const int numFrames = 600;     // Total number of frames are set to 600, more can be made for a longer animation
    const double zoomFactor = 1.02; // Zoom speed
    const char *outputDir = "frames";

    // Starting region (Seahorse Valley)
    double centerX = -0.743643887037151;
    double centerY = 0.13182590420533;
    double scale = 4.0; 

    system("mkdir -p frames");

    unsigned char *h_output = (unsigned char *)malloc(WIDTH * HEIGHT * 3);
    unsigned char *d_output;
    hipMalloc((void **)&d_output, WIDTH * HEIGHT * 3);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    printf("Generating Mandelbrot zoom animation...\n");

    for (int frame = 0; frame < numFrames; frame++) {
        double xmin = centerX - scale / 2;
        double xmax = centerX + scale / 2;
        double ymin = centerY - scale / 2;
        double ymax = centerY + scale / 2;

        computeMandelbrot<<<numBlocks, threadsPerBlock>>>(xmin, xmax, ymin, ymax, WIDTH, HEIGHT, d_output);
        hipDeviceSynchronize();

        hipMemcpy(h_output, d_output, WIDTH * HEIGHT * 3, hipMemcpyDeviceToHost);

        char filename[256];
        sprintf(filename, "%s/frame_%04d.ppm", outputDir, frame);
        savePPM(filename, h_output, WIDTH, HEIGHT);

        scale /= zoomFactor;

        printf("Frame %d/%d saved to %s\n", frame + 1, numFrames, filename);
    }

    free(h_output);
    hipFree(d_output);

    printf("All frames generated. Use ffmpeg to create a video.\n");
    return 0;
}